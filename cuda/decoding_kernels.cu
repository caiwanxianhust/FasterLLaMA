#include "hip/hip_runtime.h"
#include "decoding_kernels.cuh"

namespace tinycudallama
{
    /**
     * decoding_params.sequence_length is initialized by 0
     * finished_buf_ is initialized by false
     */
    __global__ void topKSamplingInitKernel(bool *__restrict__ finished, int *__restrict__ sequence_length, const int batch_size)
    {
        int tid = threadIdx.x;
        if (tid < batch_size)
        {
            finished[tid] = false;
            sequence_length[tid] = 0;
        }
    }

    void launchTopKSamplingInitKernel(bool *__restrict__ finished, int *__restrict__ sequence_length,
                                      const int batch_size, hipStream_t stream)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        dim3 grid(1);
        dim3 block(min(1024, batch_size));
        topKSamplingInitKernel<<<grid, block, 0, stream>>>(finished, sequence_length, batch_size);
    }

    /**
     * decoding_params.sequence_length is initialized by 0
     * finished_buf_ is initialized by false
     * topp_offset_buf is initialized by [0, vocab_size, ..., batch_size * vocab_size]
     * topp_id_val_buf is initialized by [[0, 1, ..., vocab_size-1], [0, 1, ..., vocab_size-1], ..., [0, 1, ..., vocab_size-1]]
     */
    __global__ void topPInitializationKernel(bool *__restrict__ finished, int *__restrict__ sequence_length,
                                             int *__restrict__ topp_id_val_buf, int *__restrict__ topp_offset_buf,
                                             const int batch_size, const int vocab_size)
    {
        int tid = threadIdx.x;
        int bid = blockIdx.x;

        if (bid == 0)
        {
            for (int i = tid; i < batch_size + 1; i += blockDim.x)
            {
                topp_offset_buf[i] = i * vocab_size;
            }

            for (int i = tid; i < batch_size; i += blockDim.x)
            {
                finished[i] = false;
                sequence_length[i] = 0;
            }
        }

        for (int idx = tid + bid * blockDim.x; idx < batch_size * vocab_size; idx += blockDim.x * gridDim.x)
        {
            topp_id_val_buf[idx] = idx % vocab_size;
        }
    }

    void launchTopPInitializationKernel(bool *__restrict__ finished, int *__restrict__ sequence_length,
                                        int *__restrict__ topp_id_val_buf, int *__restrict__ topp_offset_buf,
                                        const int batch_size, const int vocab_size, hipStream_t stream)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        topPInitializationKernel<<<32, 512, 0, stream>>>(finished, sequence_length, topp_id_val_buf, topp_offset_buf,
                                                         batch_size, vocab_size);
    }

    template <typename T>
    __global__ void embeddingLookupKernel(T *__restrict__ from_tensor, const T *__restrict__ embedding_table,
                                          const int *__restrict__ word_ids, const int hidden_units)
    {
        const int token_id = blockIdx.x;
        const int batch_id = blockIdx.y;
        int write_pos, lookup_pos;
        for (int tid = threadIdx.x; tid < hidden_units; tid += blockDim.x)
        {
            write_pos = tid + token_id * hidden_units + batch_id * gridDim.x * hidden_units;
            lookup_pos = word_ids[batch_id * gridDim.x + token_id] * hidden_units + tid;
            // 1. lookup the table
            // 2. multiply hidden_dim**0.5
            // if (lookup_pos < 0) {
            //     printf("batch_id: %d  token_id: %d  word_id: %d\n", batch_id, token_id, word_ids[batch_id * gridDim.x + token_id]);
            // }

            from_tensor[write_pos] = embedding_table[lookup_pos] * (T)sqrtf(float(hidden_units));
        }
    }

    template <typename T>
    void launchEmbeddingLookupKernel(T *__restrict__ from_tensor, const T *__restrict__ embedding_table, const int *__restrict__ word_ids,
                                     const int batch_size, const int cur_seq_len, const int hidden_units,
                                     hipStream_t stream)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        dim3 grid(cur_seq_len, batch_size);
        dim3 block(256);
        embeddingLookupKernel<T><<<grid, block, 0, stream>>>(from_tensor, embedding_table, word_ids, hidden_units);
    }

    /** 取 logits[:, -1, :] 存入 step_logits，并顺便进行停止符判断
     * grid(batch_size), block(min(vocab_size, 1024))
     * step_logits: [batch_size, 1, vocab_size]
     * logits: [batch_size, seq_len, vocab_size]
     * finished: [batch_size, 1]
     */
    __global__ void updateLogitsWithoutSoftmax(float *__restrict__ step_logits, const float *__restrict__ logits, const int end_id,
                                               const bool *__restrict__ finished, const int seq_len, const int vocab_size)
    {
        const bool is_finished = finished[blockIdx.x];

        for (int tid = threadIdx.x; tid < vocab_size; tid += blockDim.x)
        {
            int idx = blockIdx.x * seq_len * vocab_size + (seq_len - 1) * vocab_size + tid;
            if (is_finished)
            {
                step_logits[blockIdx.x * vocab_size + tid] = (tid == end_id) ? FLT_MAX : -1 * FLT_MAX;
            }
            else
            {
                step_logits[blockIdx.x * vocab_size + tid] = logits[idx];
            }
        }
    }

    void launchUpdateLogitsWithoutSoftmax(float *__restrict__ step_logits, const float *__restrict__ logits, const int end_id,
                                          const bool *__restrict__ finished, const int batch_size, const int seq_len,
                                          const int vocab_size, hipStream_t stream)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        dim3 grid(batch_size);
        dim3 block(min(vocab_size, 1024));
        /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big. */
        updateLogitsWithoutSoftmax<<<grid, block, 0, stream>>>(step_logits, logits, end_id, finished, seq_len, vocab_size);
    }

    /**
     * top-k Sampling kernel
     * grid(1), block(batch_size)
     */
    template <typename T>
    __global__ void topKSampling(int *__restrict__ topk_tmp_id_buf, T *__restrict__ topk_tmp_val_buf, int *__restrict__ ids,
                                 int *__restrict__ sequence_length, bool *__restrict__ finished_buf,
                                 const int *__restrict__ prompt_tokens, const bool *__restrict__ prompt_tokens_mask,
                                 const int cur_pos, const int max_prompt_seq_len, const int candidate_num,
                                 const int random_num, const int end_id, const int batch_size, const int vocab_size)
    {
        if (threadIdx.x < batch_size)
        {
            // prompt phase, next_token[:] = prompt_tokens[:, cur_pos]
            if (cur_pos < max_prompt_seq_len && prompt_tokens_mask[threadIdx.x * max_prompt_seq_len + cur_pos])
            {
                ids[threadIdx.x] = prompt_tokens[threadIdx.x * max_prompt_seq_len + cur_pos];
            }
            else
            {
                // The maximum number of k logits in the current batch
                float max_val = (float)topk_tmp_val_buf[threadIdx.x * candidate_num];

                float sum = 0.0f;
                float tmp_val;
                for (int i = 0; i < candidate_num; ++i)
                {
                    tmp_val = __expf(topk_tmp_val_buf[threadIdx.x * candidate_num + i] - max_val);
                    topk_tmp_val_buf[threadIdx.x * candidate_num + i] = tmp_val;
                    sum += tmp_val;
                }

                hiprandState_t local_state;
                hiprand_init(random_num, threadIdx.x, 0, &local_state);
                float rand_num = hiprand_uniform(&local_state) * sum;

                ids[threadIdx.x] = topk_tmp_id_buf[threadIdx.x * candidate_num + candidate_num - 1] % vocab_size;
                for (int i = 0; i < candidate_num; i++)
                {
                    rand_num = rand_num - topk_tmp_val_buf[threadIdx.x * candidate_num + i];
                    if (rand_num <= 0.0f)
                    {
                        ids[threadIdx.x] = topk_tmp_id_buf[threadIdx.x * candidate_num + i] % vocab_size;
                        break;
                    }
                }

                sequence_length[threadIdx.x] = finished_buf[threadIdx.x] ? sequence_length[threadIdx.x] : sequence_length[threadIdx.x] + 1;
                finished_buf[threadIdx.x] = ids[threadIdx.x] == end_id ? true : false;
            }
        }
    }

    template <typename T, int MAX_K, int THREADBLOCK_SIZE>
    __launch_bounds__(THREADBLOCK_SIZE)
        __global__
        void beam_topK_kernel(const T *__restrict__ log_probs,
                              int *__restrict__ topk_tmp_id_buf,
                              T *__restrict__ topk_tmp_val_buf,
                              const int vocab_size,
                              T diversity_rate)
    {
        typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
        __shared__ typename BlockReduce::TempStorage temp_storage;

        int thread_id = threadIdx.x;
        int block_id = blockIdx.x;
        TopK<T, MAX_K> partial;

#pragma unroll
        for (int i = 0; i < MAX_K; ++i)
        {
            partial.p[i] = -1;
            partial.u[i] = -FLT_MAX;
        }

#pragma unroll
        for (int elem_id = thread_id; elem_id < vocab_size; elem_id += THREADBLOCK_SIZE)
        {
            int index = elem_id + block_id * vocab_size;
            partial.insert(log_probs[index], index);
        }

        TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

        if (thread_id == 0)
        {
            int index = block_id * MAX_K;

#pragma unroll
            for (int i = 0; i < MAX_K; ++i)
            {
                topk_tmp_id_buf[index + i] = total.p[i];
                topk_tmp_val_buf[index + i] = total.u[i] + diversity_rate * (T)i;
            }
        }
    }

    template <typename T>
    void launchTopKSamplingKernel(T *__restrict__ log_probs, int *__restrict__ topk_tmp_id_buf, T *__restrict__ topk_tmp_val_buf,
                                  int *__restrict__ ids, int *__restrict__ sequence_length, bool *__restrict__ finished_buf,
                                  const int *__restrict__ prompt_tokens, const bool *__restrict__ prompt_tokens_mask,
                                  const int cur_pos, const int max_prompt_seq_len, int random_num, const int batch_size,
                                  const int vocab_size, const int candidate_num, const int end_id, hipStream_t stream)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        int local_block_size = 256;
        switch (candidate_num)
        {
            CASE_K(1);
            CASE_K(2);
            CASE_K(4);
        default:
            printf("[ERROR] Topk kernel does not support candidate_num = %d \n", candidate_num);
            exit(0);
            break;
        }
        assert(batch_size <= 1024);
        if (batch_size <= 128)
        {
            local_block_size = 128;
        }
        else if (batch_size <= 256)
        {
            local_block_size = 256;
        }
        else if (batch_size <= 512)
        {
            local_block_size = 512;
        }
        else
        {
            local_block_size = 1024;
        }
        topKSampling<T><<<1, local_block_size, 0, stream>>>(topk_tmp_id_buf, topk_tmp_val_buf, ids, sequence_length, finished_buf,
                                                            prompt_tokens, prompt_tokens_mask, cur_pos, max_prompt_seq_len, candidate_num,
                                                            random_num, end_id, batch_size, vocab_size);
    }

    __global__ void updateLogitsKernelWithoutLog(float *__restrict__ step_logits, const float *__restrict__ logits,
                                                 const bool *__restrict__ finished,
                                                 const int seq_len, const int end_id, const int vocab_size)
    {
        int bid = blockIdx.x;
        bool finish = finished[bid];
        int offset = bid * vocab_size;

        float max_val = -1 * FLT_MAX;

        for (int tid = threadIdx.x; tid < vocab_size; tid += blockDim.x)
        {
            int idx = bid * seq_len * vocab_size + (seq_len - 1) * vocab_size + tid;
            if (finish)
                step_logits[offset + tid] = (tid == end_id) ? FLT_MAX : -1 * FLT_MAX;
            else
                step_logits[offset + tid] = logits[idx];
            max_val = max(max_val, step_logits[offset + tid]);
        }

        max_val = blockAllReduceMax<float>(max_val);

        float sum_val = 0.0f;
        for (int tid = threadIdx.x; tid < vocab_size; tid += blockDim.x)
        {
            step_logits[offset + tid] = __expf(step_logits[offset + tid] - max_val);
            sum_val += step_logits[offset + tid];
        }

        sum_val = blockAllReduceSum<float>(sum_val);

        for (int tid = threadIdx.x; tid < vocab_size; tid += blockDim.x)
        {
            step_logits[offset + tid] = (step_logits[offset + tid] / sum_val);
        }
    }

    void launchUpdateLogitsKernelWithoutLog(float *__restrict__ step_logits, const float *__restrict__ logits,
                                            const bool *__restrict__ finished, const int seq_len, const int end_id,
                                            const int batch_size, const int vocab_size, hipStream_t stream)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        dim3 grid(batch_size);
        dim3 block(min(vocab_size, 1024));
        /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big. */
        updateLogitsKernelWithoutLog<<<grid, block, 0, stream>>>(step_logits, logits, finished, seq_len, end_id, vocab_size);
    }

    /**
     * top-k Sampling kernel
     * grid(1), block(batch_size)
     */
    template <typename T>
    __global__ void topPSampling(const T *__restrict__ sorted_logits_probs, const int *__restrict__ sorted_id_vals,
                                 int *__restrict__ ids, int *__restrict__ sequence_length, bool *__restrict__ finished_buf,
                                 const int *__restrict__ prompt_tokens, const bool *__restrict__ prompt_tokens_mask,
                                 const int cur_pos, const int max_prompt_seq_len, const int batch_size, const int vocab_size,
                                 const int random_num, const float prob_threshold, const int end_id)
    {
        if (threadIdx.x < batch_size)
        {
            // prompt phase, next_token[:] = prompt_tokens[:, cur_pos]
            if (cur_pos < max_prompt_seq_len && prompt_tokens_mask[threadIdx.x * max_prompt_seq_len + cur_pos])
            {
                ids[threadIdx.x] = prompt_tokens[threadIdx.x * max_prompt_seq_len + cur_pos];
            }
            else
            {
                int tid = threadIdx.x;
                hiprandState_t local_state;
                hiprand_init(random_num, tid, 0, &local_state);
                float rand_num = hiprand_uniform(&local_state) * prob_threshold;
                ids[tid] = sorted_id_vals[vocab_size - 1];

                for (int i = tid * vocab_size; i < tid * vocab_size + vocab_size; i++)
                {
                    rand_num = rand_num - sorted_logits_probs[i];
                    if (rand_num <= 0)
                    {
                        ids[tid] = sorted_id_vals[i];
                        break;
                    }
                }

                sequence_length[tid] = finished_buf[tid] ? sequence_length[tid] : sequence_length[tid] + 1;
                finished_buf[tid] = ids[tid] == end_id ? true : false;
            }
        }
    }

    /**
     * Get the temporary memory buffer size of topp sort by calling the function: hipcub::DeviceSegmentedRadixSort::SortPairsDescending
     */
    size_t getToppSortTempStorageSize(const float *__restrict__ log_probs,
                                      const int *__restrict__ id_vals,
                                      float *__restrict__ sorted_log_probs,
                                      int *__restrict__ sorted_id_vals,
                                      int *__restrict__ topp_offset_buf,
                                      const int batch_size,
                                      const int vocab_size)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;

        hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage,
                                                           temp_storage_bytes,
                                                           log_probs,
                                                           sorted_log_probs,
                                                           id_vals,
                                                           sorted_id_vals,
                                                           vocab_size * batch_size,
                                                           batch_size,
                                                           topp_offset_buf, topp_offset_buf + 1);
        return temp_storage_bytes;
    }

    template <typename T>
    void launchTopPSamplingKernel(const T *__restrict__ logits_probs, const int *__restrict__ id_vals, T *__restrict__ sorted_logits_probs,
                                  int *__restrict__ sorted_id_vals, const int *__restrict__ topp_offset_buf, void *__restrict__ temp_storage,
                                  size_t temp_storage_size, bool *__restrict__ finished_buf, const int *__restrict__ prompt_tokens,
                                  const bool *__restrict__ prompt_tokens_mask, const int cur_pos, const int max_prompt_seq_len,
                                  const int random_num, int *__restrict__ output_ids, int *__restrict__ sequence_length, const int end_id,
                                  const int batch_size, const int vocab_size, const float probability_threshold, hipStream_t stream)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage,
                                                           temp_storage_size,
                                                           logits_probs,
                                                           sorted_logits_probs,
                                                           id_vals,
                                                           sorted_id_vals,
                                                           vocab_size * batch_size,
                                                           batch_size,
                                                           topp_offset_buf, topp_offset_buf + 1);

        int local_block_size;
        assert(batch_size <= 1024);
        if (batch_size <= 128)
        {
            local_block_size = 128;
        }
        else if (batch_size <= 256)
        {
            local_block_size = 256;
        }
        else if (batch_size <= 512)
        {
            local_block_size = 512;
        }
        else
        {
            local_block_size = 1024;
        }

        topPSampling<<<1, local_block_size, 0, stream>>>(sorted_logits_probs, sorted_id_vals, output_ids, sequence_length,
                                                         finished_buf, prompt_tokens, prompt_tokens_mask, cur_pos, max_prompt_seq_len,
                                                         batch_size, vocab_size, random_num, probability_threshold, end_id);
    }

    __global__ void removePromptTokenKernel(int *__restrict__ gen_ids, const int *__restrict__ word_ids_buf,
                                            const int *__restrict__ sequence_length, const int *__restrict__ prompt_seq_lengths,
                                            const int min_prompt_seq_len, const int batch_size, const int total_len)
    {
        const int offset = prompt_seq_lengths[blockIdx.x] - min_prompt_seq_len;
        for (int tid = threadIdx.x; tid < sequence_length[blockIdx.x]; tid += blockDim.x)
        {
            gen_ids[blockIdx.x * total_len + tid] = word_ids_buf[(offset + tid) * batch_size + blockIdx.x];
            // printf("batch_id: %d tid: %d  word_id: %d\n", blockIdx.x, tid, gen_ids[blockIdx.x * total_len + tid]);
        }
    }

    void launchRemovePromptTokenKernel(int *__restrict__ gen_ids, const int *__restrict__ word_ids_buf, const int *__restrict__ sequence_length,
                                       const int *__restrict__ prompt_seq_lengths, const int min_prompt_seq_len, const int batch_size, const int total_len, hipStream_t stream)
    {
#ifndef NDEBUG
        PRINT_FUNC_NAME_();
#endif
        removePromptTokenKernel<<<batch_size, 256, 0, stream>>>(gen_ids, word_ids_buf, sequence_length, prompt_seq_lengths, min_prompt_seq_len, batch_size, total_len);
    }

    template void launchEmbeddingLookupKernel(float *__restrict__ from_tensor, const float *__restrict__ embedding_table,
                                              const int *__restrict__ word_ids, const int batch_size, const int cur_seq_len,
                                              const int hidden_units, hipStream_t stream);

    template void launchTopKSamplingKernel(float *__restrict__ log_probs, int *__restrict__ topk_tmp_id_buf,
                                           float *__restrict__ topk_tmp_val_buf, int *__restrict__ ids,
                                           int *__restrict__ sequence_length, bool *__restrict__ finished_buf,
                                           const int *__restrict__ prompt_tokens, const bool *__restrict__ prompt_tokens_mask,
                                           const int cur_pos, const int max_prompt_seq_len, int random_num, const int batch_size,
                                           const int vocab_size, const int candidate_num, const int end_id, hipStream_t stream);

    template void launchTopPSamplingKernel(const float *__restrict__ logits_probs, const int *__restrict__ id_vals,
                                           float *__restrict__ sorted_logits_probs, int *__restrict__ sorted_id_vals,
                                           const int *__restrict__ topp_offset_buf, void *__restrict__ temp_storage,
                                           size_t temp_storage_size,
                                           bool *__restrict__ finished_buf, const int *__restrict__ prompt_tokens,
                                           const bool *__restrict__ prompt_tokens_mask, const int cur_pos, const int max_prompt_seq_len,
                                           const int random_num, int *__restrict__ output_ids, int *__restrict__ sequence_length,
                                           const int end_id,
                                           const int batch_size, const int vocab_size, const float probability_threshold,
                                           hipStream_t stream);
}
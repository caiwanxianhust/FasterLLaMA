#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"

#include <cstdio>
#include <cstdlib>

template <typename T>
void printVecInVec(const T *clusters, const int nrows, const int ncols, const int end_row, const int end_col, const char *str)
{
    printf("%s:\n[\n", str);
    for (int i = 0; i < end_row; ++i)
    {
        printf("[");
        for (int j = 0; j < end_col; ++j)
        {
            printf("%g  ", static_cast<float>(clusters[i * ncols + j]));
        }
        printf("]\n");
    }
    printf("]\n");
}

template <>
void printVecInVec(const half *clusters, const int nrows, const int ncols, const int end_row, const int end_col, const char *str)
{
    printf("%s:\n[\n", str);
    if (end_row >= nrows || end_col >= ncols) printf("invalid arguments!!!\nend_row >= nrows or end_col >= ncols\n");
    for (int i = 0; i < end_row; ++i)
    {
        printf("[");
        for (int j = 0; j < end_col; ++j)
        {
            printf("%g  ", __half2float(clusters[i * ncols + j]));
        }
        printf("]\n");
    }
    printf("]\n");
}

template<typename T>
void device_malloc(T** ptr, int size)
{
    CHECK_CUDA_ERROR(hipMalloc((void**)ptr, sizeof(T) * size));
    T* tmp = new T[size];
    for(int i = 0; i < size; i++) tmp[i] = (T)((float) rand() / (RAND_MAX + 1.0) * 0.02);
    CHECK_CUDA_ERROR(hipMemcpy(*ptr, tmp, sizeof(T) * size, hipMemcpyHostToDevice));
    delete [] tmp;
}

__global__ void convertMatfloat2half(const float *input, half *output, const int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i=offset; i<size; i+=gridDim.x * blockDim.x) {
        output[i] = __float2half(input[i]);
    }
}


template <typename DataType>
void timingResNorm(DataType* output, const DataType* input, const DataType* gamma, const float eps, const int m, const int n, 
    DataType *h_out, const int method)
{
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    CHECK_CUDA_ERROR(hipEventRecord(start));
    hipEventQuery(start);

    switch (method)
    {
    case 0:
        tinycudallama::launchResNormKernel(output, input, gamma, 1e-7f, m, n);
        break;
    // case 1:
    //     tinycudallama::rms_norm_f32_cuda(input, output, n, m, gamma, 1e-7f);
    default:
        tinycudallama::launchResNormKernel(output, input, gamma, 1e-7f, m, n);
        break;
    }

    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float elapsedTime;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
    
    printf("Time = %g ms.\n", elapsedTime);

    CHECK_CUDA_ERROR(hipMemcpy(h_out, output, sizeof(DataType) * (m * n), hipMemcpyDeviceToHost));
    printf("method : %d\n", method);
    printVecInVec(h_out, m, n, 10, 10, "h_out");

}


void testResNorm()
{
    using DataType = float;
    const int m = 10000;
    const int n = 4096;
    
    DataType *h_out1 = new DataType[m * n * 2];
    DataType *h_out2 = h_out1 + m * n;

    std::srand(1234);

    DataType *d_in;
    DataType *d_out1;
    DataType *d_out2;
    DataType *d_gamma;
    device_malloc<DataType>(&d_in, sizeof(DataType) * (m * n * 3 + n));
    d_out1 = d_in + m * n;
    d_out2 = d_out1 + m * n;
    d_gamma = d_out2 + m * n;

    timingResNorm<DataType>(d_out1, d_in, d_gamma, 1e-7f, m, n, h_out1, 0);
    timingResNorm<DataType>(d_out2, d_in, d_gamma, 1e-7f, m, n, h_out2, 1);

    half *d_in_half;
    half *d_out_half;
    half *d_gamma_half;
    half *h_out_half = new half[m * n];
    
    device_malloc(&d_in_half, sizeof(half) * (m * n * 2 + n));
    d_out_half = d_in_half + m * n;
    d_gamma_half = d_out_half + m * n;

    convertMatfloat2half<<<m, 256>>>(d_in, d_in_half, m * n);

    convertMatfloat2half<<<1, 256>>>(d_gamma, d_gamma_half, n);
    timingResNorm<half>(d_out_half, d_in_half, d_gamma_half, 1e-7f, m, n, h_out_half, 0);

    CHECK_CUDA_ERROR(hipFree(d_in));
    CHECK_CUDA_ERROR(hipFree(d_in_half));
    delete [] h_out_half;
    delete [] h_out1;
}


void testPrecomputeFreqsCis()
{
    const int seq_len = 4096;
    const int size_per_head = 32;
    float *h_freqsCis = new float[seq_len * size_per_head];

    float *d_freqs_cis;

    device_malloc<float>(&d_freqs_cis, sizeof(float) * (seq_len * size_per_head));

    tinycudallama::launchPrecomputeFreqsCis(d_freqs_cis, size_per_head, seq_len);

    CHECK_CUDA_ERROR(hipMemcpy(h_freqsCis, d_freqs_cis, sizeof(float) * (seq_len * size_per_head), hipMemcpyDeviceToHost));
    printVecInVec(h_freqsCis, seq_len, size_per_head, 10, size_per_head, "freqs_cis");

    CHECK_CUDA_ERROR(hipFree(d_freqs_cis));
    delete [] h_freqsCis;
}

void testEmbedding()
{
    using DataType = float;
    const int batch_size = 2;
    const int seq_len = 4;
    const int hidden_units = 32;
    int word_ids[batch_size * seq_len] = {0, 1, 2, 3, 3, 2, 1, 0};

    DataType *h_embedding_table = new DataType[5 * hidden_units];
    DataType *h_from_tensor = new DataType[batch_size * seq_len * hidden_units];
    DataType *d_embedding_table;
    DataType *from_tensor;
    int *d_word_ids;

    device_malloc(&d_embedding_table, sizeof(DataType) * 5 * hidden_units);
    device_malloc(&from_tensor, sizeof(DataType) * batch_size * seq_len * hidden_units);
    device_malloc(&d_word_ids, sizeof(int) * batch_size * seq_len);

    CHECK_CUDA_ERROR(hipMemcpy(h_embedding_table, d_embedding_table, sizeof(DataType) * 5 * hidden_units, hipMemcpyDeviceToHost));
    printVecInVec(h_embedding_table, 5, hidden_units, 5, hidden_units, "embedding_table");

    CHECK_CUDA_ERROR(hipMemcpy(d_word_ids, word_ids, sizeof(int) * batch_size * seq_len, hipMemcpyHostToDevice));

    tinycudallama::launchEmbeddingLookingUpKernel<DataType>(from_tensor, d_embedding_table, d_word_ids, hidden_units, batch_size, seq_len);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipMemcpy(h_from_tensor, from_tensor, sizeof(DataType) * batch_size * seq_len * hidden_units, hipMemcpyDeviceToHost));
    printVecInVec(h_from_tensor, batch_size * seq_len, hidden_units, batch_size * seq_len, hidden_units, "from_tensor");

    CHECK_CUDA_ERROR(hipFree(d_embedding_table));
    CHECK_CUDA_ERROR(hipFree(from_tensor));
    CHECK_CUDA_ERROR(hipFree(d_word_ids));

    delete [] h_embedding_table;
    delete [] h_from_tensor;
}

void testPerChannelQuantized()
{
    using DataType = float;
    const int nrows = 8;
    const int hidden_size = 32;
    DataType *h_src = new DataType[nrows * hidden_size];
    for (int i=0; i<nrows*hidden_size; ++i) {
        h_src[i] = ((i & 1) == 0) ? i : (-1.0f) * i;
    }
    printVecInVec(h_src, nrows, hidden_size, nrows, hidden_size, "h_src");

    DataType *d_src;
    float *d_scale;
    int8_t *d_dst;

    device_malloc(&d_src, sizeof(DataType) * nrows * hidden_size);
    device_malloc(&d_scale, sizeof(float) * nrows);
    device_malloc(&d_dst, sizeof(int8_t) * nrows * hidden_size);

    CHECK_CUDA_ERROR(hipMemcpy(d_src, h_src, sizeof(DataType) * nrows * hidden_size, hipMemcpyHostToDevice));

    tinycudallama::perChannelQuantizedKernelLauncher(d_dst, d_src, d_scale, hidden_size, nrows);

    float h_scale[nrows];
    int8_t *h_dst = new int8_t[nrows * hidden_size];
    CHECK_CUDA_ERROR(hipMemcpy(h_scale, d_scale, sizeof(float) * nrows, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(h_dst, d_dst, sizeof(int8_t) * nrows * hidden_size, hipMemcpyDeviceToHost));

    printVecInVec(h_scale, 1, nrows, 1, nrows, "scale");
    printVecInVec(h_dst, nrows, hidden_size, nrows, hidden_size, "int8_dst");

    CHECK_CUDA_ERROR(hipFree(d_src));
    CHECK_CUDA_ERROR(hipFree(d_scale));
    CHECK_CUDA_ERROR(hipFree(d_dst));
    delete [] h_src;
    delete [] h_dst;
}


int main()
{
    // testResNorm();

    // testPrecomputeFreqsCis();

    // testEmbedding();

    testPerChannelQuantized();

    return 0;
}